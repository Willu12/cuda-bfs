#include "hip/hip_runtime.h"
#include <iostream>
#include <queue>
#include <ctime>
#include "kernels.cuh"
#include "bfs_prefix_scan.cuh"
#include "graph.hpp"
#include "hip/hip_runtime.h"
#include "scan.cuh"
#include ""
#include "bfs_layer_count.cuh"



void compute_bfs(const Graph& g, int start, int end, std::vector<int>& prev);
void cpu_BFS(const Graph& g, int start, int end);
int main(int argc, char** argv) {
    const char *path = "data/california.txt";
    int start = 120;
    int end = 1132332;
    if(argc == 4) {
        path = argv[1];
        start = atoi(argv[2]);
        end = atoi(argv[3]);
    }
    Graph new_graph = get_Graph_from_file(path);
    cpu_BFS(new_graph,start,end);
    cuda_BFS_prefix_scan(new_graph, start, end);
    cuda_BFS_frontier_numbers(new_graph,start,end);

    return 0;
}

void compute_bfs(const Graph& g, int start, int end, std::vector<int>& prev) {
    std::vector<bool> visited(g.n);
    std::queue<int> Q;

    Q.push(start);
    visited[start] = true;

    while(!Q.empty()) {
        int v = Q.front();
        Q.pop();

        if(visited[end]) break;

        int neighbours_count = g.v_adj_length[v];
        int neighbours_offset = g.v_adj_begin[v];
        for(int i =0; i<neighbours_count; i++) {
            int neighbour = g.v_adj_list[neighbours_offset + i];

            if(!visited[neighbour]) {
                visited[neighbour] = true;
                prev[neighbour] = v;
                Q.push(neighbour);

                if(neighbour == end) {
                    break;
                }
            }
        }
    }
}

void cpu_BFS(const Graph &g, int start, int end) {
    std::vector<int> prev(g.n);
    for(int v = 0; v<g.n; v++) {
        prev[v] = UINT_MAX;
    }

    double duration;
    std::clock_t start_clock = std::clock();
    //start_clock = std::clock();
    compute_bfs(g,start,end,prev);
    duration = (double) (std::clock() - start_clock) /  (double) CLOCKS_PER_SEC;

    std::cout<<"cpu bfs took: "<<duration <<" seconds\n";

    get_path(start,end,prev.data(),g.n,"cpu_output.txt");
}