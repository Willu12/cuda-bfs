#include "hip/hip_runtime.h"
#include <iostream>
#include <queue>
#include <ctime>
#include <fstream>
#include "graph.hpp"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>


void compute_bfs(const Graph& g, unsigned int start, unsigned int end, std::vector<unsigned int>& prev);
void get_path(unsigned int start, unsigned int end, const std::vector<unsigned int>& prev,unsigned int n);
void cpu_BFS(const Graph& g, unsigned int start, unsigned int end);
hipError_t cuda_init(const Graph&, unsigned int** , unsigned int** , unsigned int** ,unsigned int** ,
unsigned int** ,bool** , bool** ,unsigned int** );
hipError_t cuda_BFS_prefix_scan(const Graph& G, unsigned int start, unsigned int end);

inline hipError_t cuda_calloc( void *devPtr, size_t size );
hipError_t create_queue(bool* frontier,unsigned int** prefix_scan, unsigned int** queue,int n);
int main() {
    Graph new_graph = get_Graph_from_file("data/california.txt");
    cpu_BFS(new_graph,732,240332);

    return 0;
}

void compute_bfs(const Graph& g, unsigned int start, unsigned int end, std::vector<unsigned int>& prev) {
    std::vector<bool> visited(g.n);
    std::queue<unsigned int> Q;

    Q.push(start);
    visited[start] = true;

    while(!Q.empty()) {
        unsigned int v = Q.front();
        Q.pop();

        if(visited[end]) break;

        unsigned int neighbours_count = g.v_adj_length[v];
        unsigned int neighbours_offset = g.v_adj_begin[v];
        for(int i =0; i<neighbours_count; i++) {
            unsigned int neighbour = g.v_adj_list[neighbours_offset + i];

            if(!visited[neighbour]) {
                visited[neighbour] = true;
                prev[neighbour] = v;
                Q.push(neighbour);

                if(neighbour == end) {
                    break;
                }
            }
        }
    }
}

void get_path(unsigned int start, unsigned int end, const std::vector<unsigned int>& prev, unsigned int n) {
    unsigned int len = 1;
    std::vector<unsigned int> path(n);
    path[0] = end;
    unsigned int v = prev[end];
    while(v != start) {
        path[len++] = v;
        v = prev[v];
    }

    std::vector<unsigned int> reversed_path(len + 1);
    reversed_path[0] = start;
    for(unsigned int i = 0; i < len ; i++) {
        reversed_path[i + 1] = path[len -1  - i];
    }

    std::ofstream output("output.txt");
    for(unsigned int i =1; i <= len; i++) {
        output <<  reversed_path[i] << '\n';
    }
    output.close();
}


void cpu_BFS(const Graph &g, unsigned int start, unsigned int end) {
    std::vector<unsigned int> prev(g.n);
    for(unsigned int v = 0; v<g.n; v++) {
        prev[v] = UINT_MAX;
    }

    std::clock_t start_clock;
    double duration;
    start_clock = std::clock();
    compute_bfs(g,start,end,prev);
    duration = (double) (std::clock() - start_clock) /  (double) CLOCKS_PER_SEC;

    std::cout<<"cpu bfs took: "<<duration <<" seconds\n";

    get_path(start,end,prev,g.n);
}

hipError_t cuda_BFS_prefix_scan(const Graph& G, unsigned int start, unsigned int end) {
    //tutaj trzeba zrobić wszystko 
    
    // inicjalizuje tabilice
    unsigned int* v_adj_list;
    unsigned int* v_adj_begin;
    unsigned int* v_adj_length;
    unsigned int* queue;
    unsigned int* prev;
    unsigned int* prefix_scan;
    bool* visited;
    bool* frontier;
    hipError_t cudaStatus;

    bool still_running = true;

    cudaStatus = cuda_init(G,&v_adj_list,&v_adj_begin,&v_adj_length,&queue,&prev,&visited,&frontier,&prefix_scan);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "cuda init failed");
        goto Error;
    }

    //po przekopiowaniu danych mamy BFS
    //byśmy chcieli mieć funkcje która policzymy nam kolejke dla zadanego 
    frontier[start] = true;

    
    //main loop

    while(still_running) {
        //create queue
        create_queue(frontier,&prefix_scan,&queue,G.n);
        //memset frontier to zero
        //do BFS LAYER
        //check if finished
    }


    Error:
    //hipFree(v_adj_list);
    //hipFree(v_adj_begin);
    //hipFree(v_adj_length);
    
    return cudaStatus;
}


hipError_t cuda_init(const Graph& G, unsigned int** v_adj_list, unsigned int** v_adj_begin, unsigned int** v_adj_length,unsigned int** queue,
unsigned int** prev,bool** visited, bool** frontier,unsigned int** prefix_scan) { 

    hipError_t cudaStatus;
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&(*v_adj_list), G.m * sizeof(unsigned int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&(*v_adj_begin), G.n * sizeof(unsigned int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&(*v_adj_length), G.n * sizeof(unsigned int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = cuda_calloc((void**)&(*queue), G.n * sizeof(unsigned int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = cuda_calloc((void**)&(*prev), G.n * sizeof(unsigned int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = cuda_calloc((void**)&(*frontier), G.n * sizeof(bool));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = cuda_calloc((void**)&(*visited), G.n * sizeof(bool));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = cuda_calloc((void**)&(*prefix_scan), G.n * sizeof(unsigned int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(*v_adj_list, G.v_adj_list.data(), G.m * sizeof(unsigned int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(*v_adj_begin, G.v_adj_begin.data(), G.n * sizeof(unsigned int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(*v_adj_length, G.v_adj_length.data(), G.n * sizeof(unsigned int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }


    //prev queue visited i frontier na 0
   // cudaM




    Error:
    hipFree(*v_adj_list);
    hipFree(*v_adj_begin);
    hipFree(*v_adj_length);

    return cudaStatus;
}

inline hipError_t cuda_calloc( void *devPtr, size_t size )
{
  hipError_t err = hipMalloc( (void**)devPtr, size );
  if( err == hipSuccess ) err = hipMemset( *(void**)devPtr, 0, size );
  return err;
}

hipError_t prefix_scan(bool* frontier, unsigned int** prefix_scan, int n) {
    
    //clear previous prefix_scan
    hipError_t err = hipMemset( *(void**)*prefix_scan, 0, n * sizeof(unsigned int) );
    //create kernel
    return err;
}


hipError_t create_queue(bool* frontier,unsigned int** prefix_scan, unsigned int** queue,int n) {
    //clear previous queue
        hipError_t err = hipMemset( *(void**)*queue, 0, n * sizeof(unsigned int) );
    return err;
}