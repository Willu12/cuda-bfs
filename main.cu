#include "hip/hip_runtime.h"
#include <iostream>
#include <queue>
#include <vector>
#include <ctime>
#include <fstream>
#include "kernels.cuh"
#include "graph.hpp"
#include "hip/hip_runtime.h"
#include "scan.cuh"
#include ""
#include <stdio.h>



void compute_bfs(const Graph& g, int start, int end, std::vector<int>& prev);
void get_path(int start, int end, const std::vector<int>& prev,int n);
void cpu_BFS(const Graph& g, int start, int end);
hipError_t cuda_init(const Graph& G, int** v_adj_list, int** v_adj_begin, int** v_adj_length,int** queue,
                      int** prev,bool** visited, int** frontier,int** prefix_scan);
void cuda_free_all(int* v_adj_list, int* v_adj_begin, int* v_adj_length,int* queue,
int* prev,bool* visited, int* frontier,int* prefix_scan);
hipError_t cuda_BFS_prefix_scan(const Graph& G, int start, int end);
void cuda_prefix_queue_iter(int* v_adj_list, int* v_adj_begin, int* v_adj_length,int* queue,bool* visited,int*frontier,int* prev,int end,bool* d_running,bool* h_running,int n);
inline hipError_t cuda_calloc( void *devPtr, size_t size );
hipError_t create_queue(int* frontier,int** prefix_scan, int** queue,int n);
int main() {
    Graph new_graph = get_Graph_from_file("data/simple.txt");
    cpu_BFS(new_graph,0,3);
    cuda_BFS_prefix_scan(new_graph, 0, 3);

    return 0;
}

void compute_bfs(const Graph& g, int start, int end, std::vector<int>& prev) {
    std::vector<bool> visited(g.n);
    std::queue<int> Q;

    Q.push(start);
    visited[start] = true;

    while(!Q.empty()) {
        int v = Q.front();
        Q.pop();

        if(visited[end]) break;

        int neighbours_count = g.v_adj_length[v];
        int neighbours_offset = g.v_adj_begin[v];
        for(int i =0; i<neighbours_count; i++) {
            int neighbour = g.v_adj_list[neighbours_offset + i];

            if(!visited[neighbour]) {
                visited[neighbour] = true;
                prev[neighbour] = v;
                Q.push(neighbour);

                if(neighbour == end) {
                    break;
                }
            }
        }
    }
}

void get_path(int start, int end, const std::vector<int>& prev, int n) {
    int len = 1;
    std::vector<int> path(n);
    path[0] = end;
    int v = prev[end];
    while(v != start) {
        path[len++] = v;
        v = prev[v];
    }

    std::vector<int> reversed_path(len + 1);
    reversed_path[0] = start;
    for(int i = 0; i < len ; i++) {
        reversed_path[i + 1] = path[len -1  - i];
    }

    std::ofstream output("output.txt");
    for(int i =1; i <= len; i++) {
        output <<  reversed_path[i] << '\n';
    }
    output.close();    
}


void cpu_BFS(const Graph &g, int start, int end) {
    std::vector<int> prev(g.n);
    for(int v = 0; v<g.n; v++) {
        prev[v] = UINT_MAX;
    }

    std::clock_t start_clock;
    double duration;
    start_clock = std::clock();
    compute_bfs(g,start,end,prev);
    duration = (double) (std::clock() - start_clock) /  (double) CLOCKS_PER_SEC;

    std::cout<<"cpu bfs took: "<<duration <<" seconds\n";

    get_path(start,end,prev,g.n);
}

hipError_t cuda_BFS_prefix_scan(const Graph& G, int start, int end) {
    //tutaj trzeba zrobić wszystko 
    
    // inicjalizuje tabilice
    int* v_adj_list = nullptr;
    int* v_adj_begin = nullptr;
    int* v_adj_length = nullptr;
    int* queue = nullptr;
    int* prev = nullptr;
    int* prefix_scan = nullptr;
    bool* visited = nullptr;
    int* frontier = nullptr;
    hipError_t cudaStatus;

    bool running = true;
    bool* d_running;
    hipMalloc(&d_running,sizeof(bool));

    cudaStatus = cuda_init(G,&v_adj_list,&v_adj_begin,&v_adj_length,&queue,&prev,&visited,&frontier,&prefix_scan);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "cuda init failed");
    }

    // hipMalloc((void**)&(frontier), G.n * sizeof(int));
   //cuda_calloc(&frontier,G.n * sizeof(int));
    //po przekopiowaniu danych mamy BFS
    //byśmy chcieli mieć funkcje która policzymy nam kolejke dla zadanego 
    //frontier[0] = true;
    //cudaStatus = hipMemset((void**)&frontier, 0, G.n * sizeof(int));
    init_frontier<<<1,1>>>(frontier,start);
    int* currentQueue = (int*)malloc(sizeof(int) * G.n);
    hipMemcpy(currentQueue, queue, G.n * sizeof(int), hipMemcpyDeviceToHost);
    std::cout<<"start queue: [";
    for(int i =0; i<G.n; i++ ) {
        std::cout <<currentQueue[i] <<" ";
    }
    std::cout<< "]\n";

    //main loop

    while(running) {
        //create queue
        create_queue(frontier,&prefix_scan,&queue,G.n);
        hipMemcpy(currentQueue, queue, G.n * sizeof(int), hipMemcpyDeviceToHost);
        std::cout<<"current queue: [";
        for(int i =0; i<G.n; i++ ) {
            std::cout <<currentQueue[i] <<" ";
        }
        std::cout<< "]\n";

        //clear frontier
        cudaStatus = hipMemset((void**)&frontier, 0, G.n * sizeof(int));

       // cuda_calloc(frontier,G.n * sizeof(int));
        
        //iterate through queue
        cuda_prefix_queue_iter(v_adj_list,v_adj_begin,v_adj_length,queue,visited,frontier,prev,end,d_running,&running,G.n);

        //debug


        //running = false;
    }


    cuda_free_all(v_adj_list,v_adj_begin, v_adj_length, queue, prev, visited, frontier, prefix_scan);

    
    //printf("krzeslo");
    
    return cudaStatus;
}


hipError_t cuda_init(const Graph& G, int** v_adj_list, int** v_adj_begin, int** v_adj_length,int** queue,
int** prev,bool** visited, int** frontier,int** prefix_scan) {

    hipError_t cudaStatus;
    /*
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }
    */

    cudaStatus = hipMalloc((void**)v_adj_list, G.m * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }



    cudaStatus = hipMalloc((void**)v_adj_begin, G.n * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)v_adj_length, G.n * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = cuda_calloc(queue, (G.n + 1) * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = cuda_calloc((void**)prev, G.n * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }


    cudaStatus = cuda_calloc((void**)frontier, G.n * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
   // hipMalloc((void**)frontier,G.n * sizeof())
    cudaStatus = cuda_calloc(visited, G.n * sizeof(bool));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = cuda_calloc(prefix_scan, G.n * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(*(void**)v_adj_list, G.v_adj_list.data(), G.m * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(*(void**)v_adj_begin, G.v_adj_begin.data(), G.n * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(*(void**)v_adj_length, G.v_adj_length.data(), G.n * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    Error:
   // cuda_free_all(*v_adj_list,*v_adj_begin, *v_adj_length, *queue, *prev, *visited, *frontier, *prefix_scan);

    return cudaStatus;
}

inline hipError_t cuda_calloc( void *devPtr, size_t size )
{
  hipError_t err = hipMalloc( (void**)devPtr, size );
  if( err == hipSuccess ) err = hipMemset( *(void**)devPtr, 0, size );
  return err;
}

hipError_t cuda_prefix_scan(int* frontier, int** prefix_scan, int n) {
    
    //clear previous prefix_scan
    hipError_t err = hipMemset( *(void**)prefix_scan, 0, n * sizeof(int) );
    if(err != hipSuccess) return err;
    scan(*prefix_scan,frontier,n);

    return err;
}

void queue_from_prefix(int* prefix_scan, int* queue,int* frontier, int n) {
    int ELEMENTS_PER_BLOCK = 1024;
    int blocks = n / ELEMENTS_PER_BLOCK;
    if(blocks == 0) blocks = 1;
	///const int sharedMemArraySize = ELEMENTS_PER_BLOCK * sizeof(int);

	//int *d_sums, *d_incr;
	//hipMalloc((void **)&d_sums, blocks * sizeof(int));
	//hipMalloc((void **)&d_incr, blocks * sizeof(int));

	//prescan_large<<<blocks, THREADS_PER_BLOCK, 2 * sharedMemArraySize>>>(d_out, d_in, ELEMENTS_PER_BLOCK, d_sums);
	

	//const int sumsArrThreadsNeeded = (blocks + 1) / 2;
	//scanLargeDeviceArray(d_incr, d_sums, blocks);


    queue_from_prescan<<<blocks,512>>>(queue, prefix_scan, frontier,n);

    //hipFree(d_sums);
	//hipFree(d_incr);
}


hipError_t create_queue(int* frontier,int** prefix_scan, int** queue,int n) {
    //clear previous queue
    hipError_t err;
    if(hipSuccess != (err = hipMemset( *(void**)queue, 0, n * sizeof(int)) )) return err;
    if(hipSuccess != (err = cuda_prefix_scan(frontier,prefix_scan,n))) return err;

    queue_from_prefix(*prefix_scan,*queue,frontier,n);

    //teraz chcemy jakby 

    return err;
}

void cuda_prefix_queue_iter(int* v_adj_list, int* v_adj_begin, int* v_adj_length,int* queue,bool* visited,int*frontier,int* prev,int end,bool* d_running,bool* h_running,int n) {
    int ELEMENTS_PER_BLOCK = 1024;
    const int blocks = n / ELEMENTS_PER_BLOCK;
    bfs_cuda_prescan_iter<<<blocks,512>>>(v_adj_list,v_adj_begin,v_adj_length,queue,frontier,visited,prev,end,d_running);
    hipMemcpy(h_running, d_running, sizeof(bool), hipMemcpyDeviceToHost);
}

void cuda_free_all(int* v_adj_list, int* v_adj_begin, int* v_adj_length,int* queue,
                   int* prev,bool* visited, int* frontier,int* prefix_scan) {
    hipFree(v_adj_list);
    hipFree(v_adj_begin);
    hipFree(v_adj_length);
    hipFree(queue);
    hipFree(prev);
    hipFree(visited);
    hipFree(frontier);
    hipFree(prefix_scan);
}