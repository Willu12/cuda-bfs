#include "hip/hip_runtime.h"
#include <iostream>
#include <queue>
#include <ctime>
#include <fstream>
#include "graph.cpp"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>


void compute_bfs(const Graph& g, unsigned int start, unsigned int end, std::vector<unsigned int>& prev);
void get_path(unsigned int start, unsigned int end, const std::vector<unsigned int>& prev,unsigned int n);
void cpu_BFS(const Graph& g, unsigned int start, unsigned int end);
hipError_t cuda_BFS_prefix_scan(const Graph& G, unsigned int start, unsigned int end);

int main() {
    Graph new_graph = get_Graph_from_file("data/california.txt");
    cpu_BFS(new_graph,732,240332);

    return 0;
}

void compute_bfs(const Graph& g, unsigned int start, unsigned int end, std::vector<unsigned int>& prev) {
    std::vector<bool> visited(g.n);
    std::queue<unsigned int> Q;

    Q.push(start);
    visited[start] = true;

    while(!Q.empty()) {
        unsigned int v = Q.front();
        Q.pop();

        if(visited[end]) break;

        unsigned int neighbours_count = g.v_adj_length[v];
        unsigned int neighbours_offset = g.v_adj_begin[v];
        for(int i =0; i<neighbours_count; i++) {
            unsigned int neighbour = g.v_adj_list[neighbours_offset + i];

            if(!visited[neighbour]) {
                visited[neighbour] = true;
                prev[neighbour] = v;
                Q.push(neighbour);

                if(neighbour == end) {
                    break;
                }
            }
        }
    }
}

void get_path(unsigned int start, unsigned int end, const std::vector<unsigned int>& prev, unsigned int n) {
    unsigned int len = 1;
    std::vector<unsigned int> path(n);
    path[0] = end;
    unsigned int v = prev[end];
    while(v != start) {
        path[len++] = v;
        v = prev[v];
    }

    std::vector<unsigned int> reversed_path(len + 1);
    reversed_path[0] = start;
    for(unsigned int i = 0; i < len ; i++) {
        reversed_path[i + 1] = path[len -1  - i];
    }

    std::ofstream output("output.txt");
    for(unsigned int i =1; i <= len; i++) {
        output <<  reversed_path[i] << '\n';
    }
    output.close();
}


void cpu_BFS(const Graph &g, unsigned int start, unsigned int end) {
    std::vector<unsigned int> prev(g.n);
    for(unsigned int v = 0; v<g.n; v++) {
        prev[v] = UINT_MAX;
    }

    std::clock_t start_clock;
    double duration;
    start_clock = std::clock();
    compute_bfs(g,start,end,prev);
    duration = (double) (std::clock() - start_clock) /  (double) CLOCKS_PER_SEC;

    std::cout<<"cpu bfs took: "<<duration <<" seconds\n";

    get_path(start,end,prev,g.n);
}

hipError_t cuda_BFS_prefix_scan(const Graph& G, unsigned int start, unsigned int end) {
    //tutaj trzeba zrobić wszystko 
    // inicjalizuje tabilice
}


