#include "hip/hip_runtime.h"
#include <iostream>
#include <queue>
#include <vector>
#include <ctime>
#include <fstream>
#include "kernels.cuh"
#include "graph.hpp"
#include "hip/hip_runtime.h"
#include "scan.cuh"
#include ""
#include <stdio.h>



void compute_bfs(const Graph& g, int start, int end, std::vector<int>& prev);
void get_path(int start, int end, const std::vector<int>& prev,int n, const std::string& fileName);
void cpu_BFS(const Graph& g, int start, int end);
hipError_t cuda_init(const Graph& G, int** v_adj_list, int** v_adj_begin, int** v_adj_length,int** queue,
                      int** prev,bool** visited, int** frontier,int** prefix_scan);
void cuda_free_all(int* v_adj_list, int* v_adj_begin, int* v_adj_length,int* queue,
int* prev,bool* visited, int* frontier,int* prefix_scan);
hipError_t cuda_BFS_prefix_scan(const Graph& G, int start, int end);
void cuda_prefix_queue_iter(int* v_adj_list, int* v_adj_begin, int* v_adj_length,int* queue,bool* visited,int*frontier,int* prev,int end,bool* d_running,bool* h_running);
inline hipError_t cuda_calloc( void *devPtr, size_t size );
hipError_t create_queue(int* frontier,int** prefix_scan, int** queue,int n);
int main() {
    Graph new_graph = get_Graph_from_file("data/california.txt");
    cpu_BFS(new_graph,0,1433232);
    cuda_BFS_prefix_scan(new_graph, 0, 1433232);

    return 0;
}

void compute_bfs(const Graph& g, int start, int end, std::vector<int>& prev) {
    std::vector<bool> visited(g.n);
    std::queue<int> Q;

    Q.push(start);
    visited[start] = true;

    while(!Q.empty()) {
        int v = Q.front();
        Q.pop();

        if(visited[end]) break;

        int neighbours_count = g.v_adj_length[v];
        int neighbours_offset = g.v_adj_begin[v];
        for(int i =0; i<neighbours_count; i++) {
            int neighbour = g.v_adj_list[neighbours_offset + i];

            if(!visited[neighbour]) {
                visited[neighbour] = true;
                prev[neighbour] = v;
                Q.push(neighbour);

                if(neighbour == end) {
                    break;
                }
            }
        }
    }
}

void get_path(int start, int end, int *prev, int n,const std::string& fileName) {
    int len = 1;
    std::vector<int> path(n);
    path[0] = end;
    int v = prev[end];
    while(v != start) {
        path[len++] = v;
        v = prev[v];
    }

    std::vector<int> reversed_path(len + 1);
    reversed_path[0] = start;
    for(int i = 0; i < len ; i++) {
        reversed_path[i + 1] = path[len -1  - i];
    }

    std::ofstream output(fileName);
    for(int i =1; i <= len; i++) {
        output <<  reversed_path[i] << '\n';
    }
    output.close();    
}


void cpu_BFS(const Graph &g, int start, int end) {
    std::vector<int> prev(g.n);
    for(int v = 0; v<g.n; v++) {
        prev[v] = UINT_MAX;
    }

    double duration;
    std::clock_t start_clock = std::clock();
    //start_clock = std::clock();
    compute_bfs(g,start,end,prev);
    duration = (double) (std::clock() - start_clock) /  (double) CLOCKS_PER_SEC;

    std::cout<<"cpu bfs took: "<<duration <<" seconds\n";

    get_path(start,end,prev.data(),g.n,"cpu_output.txt");
}

hipError_t cuda_BFS_prefix_scan(const Graph& G, int start, int end) {
    int* v_adj_list = nullptr;
    int* v_adj_begin = nullptr;
    int* v_adj_length = nullptr;
    int* queue = nullptr;
    int* prev = nullptr;
    int* prefix_scan = nullptr;
    bool* visited = nullptr;
    int* frontier = nullptr;
    hipError_t cudaStatus;

    double duration;
    std::clock_t start_clock = std::clock();

    bool stop = false;
    bool* d_stop;
    hipMalloc(&d_stop,sizeof(bool));

    cudaStatus = cuda_init(G,&v_adj_list,&v_adj_begin,&v_adj_length,&queue,&prev,&visited,&frontier,&prefix_scan);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "cuda init failed");
    }
    init_frontier<<<1,1>>>(frontier,start);

    //main loop
    while(!stop) {
        //create queue
        create_queue(frontier,&prefix_scan,&queue,G.n);
        //clear frontier
        cudaStatus = hipMemset(frontier, 0, G.n * sizeof(int));
        //bfs layer scan
        cuda_prefix_queue_iter(v_adj_list,v_adj_begin,v_adj_length,queue,visited,frontier,prev,end,d_stop,&stop);
    }

    //copy prev array to cpu
    int* h_prev = (int*)malloc(G.n * sizeof(int));
    hipMemcpy(h_prev,prev,G.n * sizeof(int),hipMemcpyDeviceToHost);
    cuda_free_all(v_adj_list,v_adj_begin, v_adj_length, queue, prev, visited, frontier, prefix_scan);

    duration = (double) (std::clock() - start_clock) /  (double) CLOCKS_PER_SEC;
    std::cout<<"gpu bfs with prefix_scan took: "<<duration <<" seconds\n";


    get_path(start,end,h_prev,G.n,"gpu_output.txt");

    return cudaStatus;
}


hipError_t cuda_init(const Graph& G, int** v_adj_list, int** v_adj_begin, int** v_adj_length,int** queue,
int** prev,bool** visited, int** frontier,int** prefix_scan) {

    hipError_t cudaStatus;
    /*
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }
    */

    cudaStatus = hipMalloc((void**)v_adj_list, G.m * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }



    cudaStatus = hipMalloc((void**)v_adj_begin, G.n * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)v_adj_length, G.n * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = cuda_calloc(queue, (G.n + 1) * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = cuda_calloc((void**)prev, G.n * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }


    cudaStatus = cuda_calloc((void**)frontier, G.n * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
   // hipMalloc((void**)frontier,G.n * sizeof())
    cudaStatus = cuda_calloc(visited, G.n * sizeof(bool));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = cuda_calloc(prefix_scan, G.n * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(*(void**)v_adj_list, G.v_adj_list.data(), G.m * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(*(void**)v_adj_begin, G.v_adj_begin.data(), G.n * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(*(void**)v_adj_length, G.v_adj_length.data(), G.n * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    Error:
   // cuda_free_all(*v_adj_list,*v_adj_begin, *v_adj_length, *queue, *prev, *visited, *frontier, *prefix_scan);

    return cudaStatus;
}

inline hipError_t cuda_calloc( void *devPtr, size_t size )
{
  hipError_t err = hipMalloc( (void**)devPtr, size );
  if( err == hipSuccess ) err = hipMemset( *(void**)devPtr, 0, size );
  return err;
}

hipError_t cuda_prefix_scan(int* frontier, int** prefix_scan, int n) {
    
    //clear previous prefix_scan

    hipError_t err = hipMemset( *(void**)prefix_scan, 0, n * sizeof(int) );

    if(err != hipSuccess) return err;

    scan(*prefix_scan,frontier,n);

    return err;
}

void queue_from_prefix(int* prefix_scan, int* queue,int* frontier, int n) {
    int ELEMENTS_PER_BLOCK = 1024;
    int blocks = n / ELEMENTS_PER_BLOCK;
    if(blocks == 0) blocks = 1;
	///const int sharedMemArraySize = ELEMENTS_PER_BLOCK * sizeof(int);

	//int *d_sums, *d_incr;
	//hipMalloc((void **)&d_sums, blocks * sizeof(int));
	//hipMalloc((void **)&d_incr, blocks * sizeof(int));

	//preScan_large<<<blocks, THREADS_PER_BLOCK, 2 * sharedMemArraySize>>>(d_out, d_in, ELEMENTS_PER_BLOCK, d_sums);
	

	//const int sumsArrThreadsNeeded = (blocks + 1) / 2;
	//scanLargeDeviceArray(d_incr, d_sums, blocks);

    queue_from_prescan<<<blocks,512>>>(queue, prefix_scan, frontier,n);

    //hipFree(d_sums);
	//hipFree(d_incr);
}


hipError_t create_queue(int* frontier,int** prefix_scan, int** queue,int n) {
    //clear previous queue
    hipError_t err;

    if(hipSuccess != (err = hipMemset( *(void**)queue, 0, n * sizeof(int)) )) return err;

    if(hipSuccess != (err = cuda_prefix_scan(frontier,prefix_scan,n))) return err;

    queue_from_prefix(*prefix_scan,*queue,frontier,n);
    return err;
}

void cuda_prefix_queue_iter(int* v_adj_list, int* v_adj_begin, int* v_adj_length,int* queue,bool* visited,int*frontier,int* prev,int end,bool* d_stop,bool* h_stop) {
    //get amount of vertices you have to iterate
    const int ELEMENTS_PER_BLOCK = 512;
    int queue_length = 0;

    hipMemcpy(&queue_length,queue,sizeof(int),hipMemcpyDeviceToHost);
    if(queue_length == 0) {
        *h_stop = true;
        return;
    }
    int blocks = queue_length / ELEMENTS_PER_BLOCK;
    int remainder = queue_length - blocks * ELEMENTS_PER_BLOCK;


    bfs_cuda_prescan_iter<<<blocks,ELEMENTS_PER_BLOCK>>>(v_adj_list,v_adj_begin,v_adj_length,queue,frontier,visited,prev,end,d_stop,0);
    bfs_cuda_prescan_iter<<<1,remainder>>>(v_adj_list,v_adj_begin,v_adj_length,queue,frontier,visited,prev,end,d_stop,blocks * ELEMENTS_PER_BLOCK);
    hipMemcpy(h_stop, d_stop, sizeof(bool), hipMemcpyDeviceToHost);
}

void cuda_free_all(int* v_adj_list, int* v_adj_begin, int* v_adj_length,int* queue,
                   int* prev,bool* visited, int* frontier,int* prefix_scan) {
    hipFree(v_adj_list);
    hipFree(v_adj_begin);
    hipFree(v_adj_length);
    hipFree(queue);
    hipFree(prev);
    hipFree(visited);
    hipFree(frontier);
    hipFree(prefix_scan);
}