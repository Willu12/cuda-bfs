#include "hip/hip_runtime.h"
#include <stdlib.h>
#include "hip/hip_runtime.h"
#include ""

#include "kernels.cuh"
#include "scan.cuh"

#define checkCudaError(o, l) _checkCudaError(o, l, __func__)

int THREADS_PER_BLOCK = 512;
int ELEMENTS_PER_BLOCK = THREADS_PER_BLOCK * 2;

int nextPowerOfTwo(int x) {
	int power = 1;
	while (power < x) {
		power *= 2;
	}
	return power;
}

float blockscan(int *output, int *input, int length){
	int *d_out, *d_in;
	const int arraySize = length * sizeof(int);

	hipMalloc((void **)&d_out, arraySize);
	hipMalloc((void **)&d_in, arraySize);
	hipMemcpy(d_out, output, arraySize, hipMemcpyHostToDevice);
	hipMemcpy(d_in, input, arraySize, hipMemcpyHostToDevice);

	// start timer
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);

	int powerOfTwo = nextPowerOfTwo(length);
	prescan_arbitrary<<<1, (length + 1) / 2, 2 * powerOfTwo * sizeof(int)>>>(d_out, d_in, length, powerOfTwo);

	// end timer
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float elapsedTime = 0;
	hipEventElapsedTime(&elapsedTime, start, stop);

	hipMemcpy(output, d_out, arraySize, hipMemcpyDeviceToHost);

	hipFree(d_out);
	hipFree(d_in);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	return elapsedTime;
}

float scan(int *output, int *input, int length) {

	// start timer
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);

	if (length > ELEMENTS_PER_BLOCK) {
		scanLargeDeviceArray(output, input, length);
	}
	else {
		scanSmallDeviceArray(output, input, length);
	}

	// end timer
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float elapsedTime = 0;
	hipEventElapsedTime(&elapsedTime, start, stop);

    
	hipEventDestroy(start);
	hipEventDestroy(stop);

	return elapsedTime;
}


void scanLargeDeviceArray(int *d_out, int *d_in, int length) {
	int remainder = length % (ELEMENTS_PER_BLOCK);
	if (remainder == 0) {
		scanLargeEvenDeviceArray(d_out, d_in, length);
	}
	else {
		// perform a large scan on a compatible multiple of elements
		int lengthMultiple = length - remainder;
		scanLargeEvenDeviceArray(d_out, d_in, lengthMultiple);

		// scan the remaining elements and add the (inclusive) last element of the large scan to this
		int *startOfOutputArray = &(d_out[lengthMultiple]);
		scanSmallDeviceArray(startOfOutputArray, &(d_in[lengthMultiple]), remainder);

		add<<<1, remainder>>>(startOfOutputArray, remainder, &(d_in[lengthMultiple - 1]), &(d_out[lengthMultiple - 1]));
	}
}

void scanSmallDeviceArray(int *d_out, int *d_in, int length) {
	int powerOfTwo = nextPowerOfTwo(length);
	prescan_arbitrary <<<1, (length + 1) / 2, 2 * powerOfTwo * sizeof(int) >>>(d_out, d_in, length, powerOfTwo);
}

void scanLargeEvenDeviceArray(int *d_out, int *d_in, int length) {
	const int blocks = length / ELEMENTS_PER_BLOCK;
	const int sharedMemArraySize = ELEMENTS_PER_BLOCK * sizeof(int);

	int *d_sums, *d_incr;
	hipMalloc((void **)&d_sums, blocks * sizeof(int));
	hipMalloc((void **)&d_incr, blocks * sizeof(int));

	prescan_large<<<blocks, THREADS_PER_BLOCK, 2 * sharedMemArraySize>>>(d_out, d_in, ELEMENTS_PER_BLOCK, d_sums);
	

	const int sumsArrThreadsNeeded = (blocks + 1) / 2;
	scanLargeDeviceArray(d_incr, d_sums, blocks);

	add<<<blocks, ELEMENTS_PER_BLOCK>>>(d_out, ELEMENTS_PER_BLOCK, d_incr);

	hipFree(d_sums);
	hipFree(d_incr);
}