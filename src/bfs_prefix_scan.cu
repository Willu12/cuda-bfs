#include "hip/hip_runtime.h"
#include "bfs_prefix_scan.cuh"

hipError_t cuda_init(const Graph& G, int** v_adj_list, int** v_adj_begin, int** v_adj_length,int** queue,
                      int** prev,bool** visited, int** frontier,int** prefix_scan) {

    hipError_t cudaStatus;
    /*
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }
    */

    cudaStatus = hipMalloc((void**)v_adj_list, G.m * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)v_adj_begin, G.n * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)v_adj_length, G.n * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = cuda_calloc(queue, (G.n + 1) * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = cuda_calloc((void**)prev, G.n * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = cuda_calloc((void**)frontier, G.n * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = cuda_calloc(visited, G.n * sizeof(bool));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = cuda_calloc(prefix_scan, G.n * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(*(void**)v_adj_list, G.v_adj_list.data(), G.m * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(*(void**)v_adj_begin, G.v_adj_begin.data(), G.n * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(*(void**)v_adj_length, G.v_adj_length.data(), G.n * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    Error:
    // cuda_free_all(*v_adj_list,*v_adj_begin, *v_adj_length, *queue, *prev, *visited, *frontier, *prefix_scan);

    return cudaStatus;
}

inline hipError_t cuda_calloc( void *devPtr, size_t size ) {
    hipError_t err = hipMalloc( (void**)devPtr, size );
    if( err == hipSuccess ) err = hipMemset( *(void**)devPtr, 0, size );
    return err;
}

hipError_t cuda_prefix_scan(int* frontier, int** prefix_scan, int n) {
    hipError_t err = hipMemset( *(void**)prefix_scan, 0, n * sizeof(int) );
    if(err != hipSuccess) return err;
    scan(*prefix_scan,frontier,n);
    return err;
}

void queue_from_prefix(int* prefix_scan, int* queue,int* frontier, int n) {
    int ELEMENTS_PER_BLOCK = 1024;
    int blocks = n / ELEMENTS_PER_BLOCK;
    if(blocks == 0) blocks = 1;
    queue_from_prescan<<<blocks,512>>>(queue, prefix_scan, frontier,n);
}


hipError_t create_queue(int* frontier,int** prefix_scan, int** queue,int n) {
    //clear previous queue
    hipError_t err;

    if(hipSuccess != (err = hipMemset( *(void**)queue, 0, n * sizeof(int)) )) return err;

    if(hipSuccess != (err = cuda_prefix_scan(frontier,prefix_scan,n))) return err;

    queue_from_prefix(*prefix_scan,*queue,frontier,n);
    return err;
}

void cuda_prefix_queue_iter(int* v_adj_list, int* v_adj_begin, int* v_adj_length,int* queue,bool* visited,int*frontier,int* prev,int end,bool* d_stop,bool* h_stop) {
    //get amount of vertices you have to iterate
    const int ELEMENTS_PER_BLOCK = 512;
    int queue_length = 0;

    hipMemcpy(&queue_length,queue,sizeof(int),hipMemcpyDeviceToHost);
    if(queue_length == 0) {
        *h_stop = true;
        return;
    }
    int blocks = queue_length / ELEMENTS_PER_BLOCK;
    int remainder = queue_length - blocks * ELEMENTS_PER_BLOCK;


    bfs_cuda_prescan_iter<<<blocks,ELEMENTS_PER_BLOCK>>>(v_adj_list,v_adj_begin,v_adj_length,queue,frontier,visited,prev,end,d_stop,0);
    bfs_cuda_prescan_iter<<<1,remainder>>>(v_adj_list,v_adj_begin,v_adj_length,queue,frontier,visited,prev,end,d_stop,blocks * ELEMENTS_PER_BLOCK);
    hipMemcpy(h_stop, d_stop, sizeof(bool), hipMemcpyDeviceToHost);
}

void cuda_free_all(int* v_adj_list, int* v_adj_begin, int* v_adj_length,int* queue,
                   int* prev,bool* visited, int* frontier,int* prefix_scan) {
    hipFree(v_adj_list);
    hipFree(v_adj_begin);
    hipFree(v_adj_length);
    hipFree(queue);
    hipFree(prev);
    hipFree(visited);
    hipFree(frontier);
    hipFree(prefix_scan);
}